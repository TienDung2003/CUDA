#include <stdio.h>

// CUDA Kernel function to add the elements of two arrays on the GPU
_global_ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    // define host variables
    int a, b, c;
    // define device variables
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    // Allocate space for device variables
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Setup input values  
    a = 2;
    b = 7;

    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    printf("The sum is %d\n",c);

    return 0;
}
